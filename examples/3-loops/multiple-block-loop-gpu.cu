
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void loop()
{
    /*
     * This idiomatic expression gives each thread
     * a unique index within the entire grid.
     */

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    printf("%d\n", i);
}

int main()
{
    loop<<<5, 5>>>();
    hipDeviceSynchronize();
}
